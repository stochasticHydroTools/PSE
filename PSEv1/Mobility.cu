#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2009-2014 The Regents of
the University of Michigan All rights reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: joaander
// Modified by Andrew Fiore


#include "Mobility.cuh"
#include "Helper.cuh"

#include "hoomd/TextureTools.h"

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

//! command to convert floats or doubles to integers
#ifdef SINGLE_PRECISION
#define __scalar2int_rd __float2int_rd
#else
#define __scalar2int_rd __double2int_rd
#endif


/*! \file Mobility.cu
    \brief Defines GPU kernel code for Mobility calculations.
*/

//! Shared memory array for partial sum of dot product kernel
extern __shared__ Scalar partial_sum[];
extern __shared__ Scalar4 shared_Fpos[];

//! Texture for reading table values
scalar4_tex_t tables1_tex;
//! Texture for reading particle positions
scalar4_tex_t pos_tex;

//! Spread particle quantities to the grid ( ALL PARTICLES SAME SIZE ) -- give one block per particle
/*! \param d_pos            positions of the particles, actually they are fetched on texture memory
    \param d_net_force      net forces on the particles
    \param gridX            x-component of force moments projected onto grid
    \param gridY            y-component of force moments projected onto grid
    \param gridZ            z-component of force moments projected onto grid
    \param group_size       size of the group, i.e. number of particles
    \param Nx               number of grid nodes in x direction
    \param Ny               number of grid nodes in y direction
    \param Nz               number of grid nodes in z direction
    \param d_group_members  index array to global HOOMD tag on each particle
    \param box              array containing box dimensions
    \param P                number of grid nodes in support of spreading Gaussians
    \param gridh            space between grid nodes in each dimension
    \param xi               Ewald splitting parameter
    \param eta              Spectral splitting parameter
    \param prefac      Spreading function prefactor
    \param expfac      Spreading function exponential factor

    One 3-D block of threads is launched per particle (block dimension = PxPxP). Max dimension
    is 10x10x10. If P > 10, each thread will do more than one grid point worth of work. 

*/
__global__ void gpu_stokes_Spread_kernel(   
        Scalar4 *d_pos,
            Scalar4 *d_net_force,
            CUFFTCOMPLEX *gridX,
            CUFFTCOMPLEX *gridY,
            CUFFTCOMPLEX *gridZ,
            int group_size,
            int Nx,
            int Ny,
            int Nz,
            unsigned int *d_group_members,
            BoxDim box,
            const int P,
            Scalar3 gridh,
            Scalar xi,
            Scalar eta,
        Scalar prefac,
        Scalar expfac 
        ){

  // Shared memory for particle force and position, so that each block
  // only has to read once
  __shared__ Scalar3 shared[2]; // 16 kb max
  
  Scalar3 *force_shared = shared;
  Scalar3 *pos_shared = &shared[1];

  // Offset for the block (i.e. particle ID within group)  
  int group_idx = blockIdx.x;

  // Offset for the thread (i.e. grid point ID within particle's support)
  int thread_offset = threadIdx.z + threadIdx.y * blockDim.z + threadIdx.x * blockDim.z*blockDim.y;
  
  // Global particle ID
  unsigned int idx = d_group_members[group_idx];
  
  // Initialize shared memory and get particle position
  if ( thread_offset == 0 ){
    Scalar4 tpos = texFetchScalar4(d_pos, pos_tex, idx);
    pos_shared[0].x = tpos.x; 
    pos_shared[0].y = tpos.y; 
    pos_shared[0].z = tpos.z;
    
    Scalar4 tforce = d_net_force[idx];
    force_shared[0].x = tforce.x;
    force_shared[0].y = tforce.y;
    force_shared[0].z = tforce.z;
  }
  __syncthreads();
  
  // Box dimension
  Scalar3 L = box.getL();
  Scalar3 Ld2 = L / 2.0;
  
  // Retrieve position from shared memory
  Scalar3 pos = pos_shared[0];
  Scalar3 force = force_shared[0];
  
  // Fractional position within box 
  Scalar3 pos_frac = box.makeFraction(pos);
  
  pos_frac.x *= (Scalar)Nx;
  pos_frac.y *= (Scalar)Ny;
  pos_frac.z *= (Scalar)Nz;
  
  // Grid index of floor of fractional position
  int x = int( pos_frac.x );
  int y = int( pos_frac.y );
  int z = int( pos_frac.z );

  // Amount of work needed for each thread to cover support
  // (Required in case support size is larger than grid dimension,
  //  but in most cases, should have n.x = n.y = n.z = 1 )
  int3 n, t;
        n.x = ( P + blockDim.x - 1 ) / blockDim.x; // ceiling
        n.y = ( P + blockDim.y - 1 ) / blockDim.y;
        n.z = ( P + blockDim.z - 1 ) / blockDim.z;

  // Grid point associated with current thread
  int Pd2 = P/2; // integer division does floor

  for ( int ii = 0; ii < n.x; ++ii ){

    t.x = threadIdx.x + ii*blockDim.x;

    for ( int jj = 0; jj < n.y; ++jj ){

      t.y = threadIdx.y + jj*blockDim.y;

      for ( int kk = 0; kk < n.z; ++kk ){

        t.z = threadIdx.z + kk*blockDim.z;

        if ( ( t.x < P ) && ( t.y < P ) && ( t.z < P ) ){

          // x,y,z indices for current thread
          // 
          // Arithmetic with P makes sure distribution is centered on the particle
          int x_inp = x + t.x - Pd2 + 1 - (P % 2) * ( pos_frac.x - Scalar( x ) < 0.5  );
          int y_inp = y + t.y - Pd2 + 1 - (P % 2) * ( pos_frac.y - Scalar( y ) < 0.5  );
          int z_inp = z + t.z - Pd2 + 1 - (P % 2) * ( pos_frac.z - Scalar( z ) < 0.5  );

          // Periodic wrapping of grid point
          x_inp = (x_inp<0) ? x_inp+Nx : ( (x_inp>Nx-1) ? x_inp-Nx : x_inp );
          y_inp = (y_inp<0) ? y_inp+Ny : ( (y_inp>Ny-1) ? y_inp-Ny : y_inp );
          z_inp = (z_inp<0) ? z_inp+Nz : ( (z_inp>Nz-1) ? z_inp-Nz : z_inp );
          
          // x,y,z coordinates for current thread
          Scalar3 pos_grid;
          pos_grid.x = gridh.x*x_inp - Ld2.x;
          pos_grid.y = gridh.y*y_inp - Ld2.y;
          pos_grid.z = gridh.z*z_inp - Ld2.z;

          // Shear the grid position 
          // !!! This only works for linear shear where the shear gradient is along y
          //     and the shear direction is along x
          pos_grid.x = pos_grid.x + box.getTiltFactorXY() * pos_grid.y;
          
          // Global index for current grid point
          int grid_idx = x_inp * Ny * Nz + y_inp * Nz + z_inp;
          
          // Distance from particle to grid node
          Scalar3 r = pos_grid - pos;
          r = box.minImage(r);
          Scalar rsq = r.x*r.x + r.y*r.y + r.z*r.z;
          
          // Magnitude of the force contribution to the current grid node
          Scalar3 force_inp = prefac * expf( -expfac * rsq ) * force;
          
          // Add force to the grid
          atomicAdd( &(gridX[grid_idx].x), force_inp.x);
          atomicAdd( &(gridY[grid_idx].x), force_inp.y);
          atomicAdd( &(gridZ[grid_idx].x), force_inp.z);
        }// check thread is within support
      }// kk
    }// jj
  }// ii

}

//! Compute the velocity from the force moments on the grid (Same Size Particles)
//
//  This is the operator "B" from the paper
//
/*! \param gridX            x-component of force moments projected onto grid
    \param gridY            y-component of force moments projected onto grid
    \param gridZ            z-component of force moments projected onto grid
    \param gridk            wave vector and scaling factor associated with each reciprocal grid node
    \param NxNyNz           total number of grid nodes
*/
__global__ void gpu_stokes_Green_kernel(
        CUFFTCOMPLEX *gridX, 
        CUFFTCOMPLEX *gridY, 
        CUFFTCOMPLEX *gridZ, 
        Scalar4 *gridk, 
        unsigned int NxNyNz
        ) {

  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  
  if ( tid < NxNyNz ) {
  
    // Read the FFT force from global memory
          Scalar2 fX, fY, fZ;
    fX.x = gridX[tid].x;  
    fX.y = gridX[tid].y;  
    fY.x = gridY[tid].x;
    fY.y = gridY[tid].y;
    fZ.x = gridZ[tid].x;
    fZ.y = gridZ[tid].y;
  
    // Current wave-space vector 
    Scalar4 tk = gridk[tid];
    Scalar ksq = tk.x*tk.x + tk.y*tk.y + tk.z*tk.z;
    Scalar k = sqrtf( ksq );
  
    // Dot product of the wave-vector with the force 
    Scalar2 kdF = (tid==0) ? make_scalar2(0.0,0.0) : make_scalar2( ( tk.x*fX.x + tk.y*fY.x + tk.z*fZ.x ) / ksq,  ( tk.x*fX.y + tk.y*fY.y + tk.z*fZ.y ) / ksq );
  
    // Scaling factor
    Scalar B = (tid==0) ? 0.0 : tk.w * ( sinf( k ) / k ) * ( sinf( k ) / k );
  
    // Write the velocity to global memory
          Scalar2 gX, gY, gZ;
    gX = make_scalar2( ( fX.x - tk.x * kdF.x ) * B, ( fX.y - tk.x * kdF.y ) * B );
    gridX[tid].x = gX.x;
    gridX[tid].y = gX.y;
    gY = make_scalar2( ( fY.x - tk.y * kdF.x ) * B, ( fY.y - tk.y * kdF.y ) * B );
    gridY[tid].x = gY.x;
    gridY[tid].y = gY.y;
    gZ = make_scalar2( ( fZ.x - tk.z * kdF.x ) * B, ( fZ.y - tk.z * kdF.y ) * B );
    gridZ[tid].x = gZ.x;
    gridZ[tid].y = gZ.y;
  }
}

//! Add velocity from grid to particles ( Same Size Particles, Block Per Particle (support) )
/*! \param d_pos            positions of the particles, actually they are fetched on texture memory
    \param d_net_force      net forces on the particles
    \param d_vel            particle velocity
    \param gridX            x-component of force moments projected onto grid
    \param gridY            y-component of force moments projected onto grid
    \param gridZ            z-component of force moments projected onto grid
    \param group_size       size of the group, i.e. number of particles
    \param Nx               number of grid nodes in x direction
    \param Ny               number of grid nodes in y direction
    \param Nz               number of grid nodes in z direction
    \param xi               Ewald splitting parameter
    \param eta              Spectral splitting parameter
    \param d_group_members  index array to global HOOMD tag on each particle
    \param box              array containing box dimensions
    \param P                number of grid nodes in support of spreading Gaussians
    \param gridh            space between grid nodes in each dimension
    \param prefac      Spreading function prefactor
    \param expfac      Spreading function exponential factor

    One 3-D block of threads is launched per particle (block dimension = PxPxP). Max dimension
    is 10x10x10 because of shared memory limitations. If P > 10, each thread will do more 
    than one grid point worth of work. 
*/
__global__ void gpu_stokes_Contract_kernel(   
          Scalar4 *d_pos,
           Scalar4 *d_vel,
           CUFFTCOMPLEX *gridX,
           CUFFTCOMPLEX *gridY,
           CUFFTCOMPLEX *gridZ,
           int group_size,
           int Nx,
           int Ny,
           int Nz,
           Scalar xi,
           Scalar eta,
           unsigned int *d_group_members,
           BoxDim box,
           const int P,
           Scalar3 gridh,
           Scalar prefac,
           Scalar expfac 
          ){

  // Shared memory for particle velocity and position, so that each block
  // only has to read one
  extern __shared__ Scalar3 shared[];
  
  Scalar3 *velocity = shared;
  Scalar3 *pos_shared = &shared[blockDim.x*blockDim.y*blockDim.z];
  
  // Particle index within each group (block per particle)
  int group_idx = blockIdx.x;

  // Thread index within the block (grid point index)
  int thread_offset = threadIdx.z + threadIdx.y * blockDim.z + threadIdx.x * blockDim.z*blockDim.y;

  // Total number of threads within the block
  int block_size = blockDim.x * blockDim.y * blockDim.z;
  
  // Global particle ID
  unsigned int idx = d_group_members[group_idx];
  
  // Initialize shared memory and get particle position
  velocity[thread_offset] = make_scalar3(0.0,0.0,0.0);
  if ( thread_offset == 0 ){
    Scalar4 tpos = texFetchScalar4(d_pos, pos_tex, idx);
    pos_shared[0] = make_scalar3( tpos.x, tpos.y, tpos.z ); 
  }
  __syncthreads();
  
  // Box dimension
  Scalar3 L = box.getL();
  Scalar3 Ld2 = L / 2.0;
  
  // Retrieve position from shared memory
  Scalar3 pos = pos_shared[0];
  
  // Fractional position within box 
  Scalar3 pos_frac = box.makeFraction(pos);
  
  pos_frac.x *= (Scalar)Nx;
  pos_frac.y *= (Scalar)Ny;
  pos_frac.z *= (Scalar)Nz;
  
  int x = int( pos_frac.x );
  int y = int( pos_frac.y );
  int z = int( pos_frac.z );
  
  // Amount of work needed for each thread to cover support
  // (Required in case support size is larger than grid dimension,
  //  but in most cases, should have n.x = n.y = n.z = 1 )
  int3 n, t;
        n.x = ( P + blockDim.x - 1 ) / blockDim.x; // ceiling
        n.y = ( P + blockDim.y - 1 ) / blockDim.y;
        n.z = ( P + blockDim.z - 1 ) / blockDim.z;
 
  // Grid point associated with current thread
  int Pd2 = P / 2; // integer division does floor
  
  for ( int ii = 0; ii < n.x; ++ii ){

    t.x = threadIdx.x + ii*blockDim.x;

    for ( int jj = 0; jj < n.y; ++jj ){

      t.y = threadIdx.y + jj*blockDim.y;

      for ( int kk = 0; kk < n.z; ++kk ){

        t.z = threadIdx.z + kk*blockDim.z;

        if( ( t.x < P ) && ( t.y < P ) && ( t.z < P ) ){

          // x,y,z indices for current thread
          // 
          // Arithmetic with P makes sure distribution is centered on the particle
          int x_inp = x + t.x - Pd2 + 1 - (P % 2) * ( pos_frac.x - Scalar( x ) < 0.5  );
          int y_inp = y + t.y - Pd2 + 1 - (P % 2) * ( pos_frac.y - Scalar( y ) < 0.5  );
          int z_inp = z + t.z - Pd2 + 1 - (P % 2) * ( pos_frac.z - Scalar( z ) < 0.5  );
          
          // Periodic wrapping of grid point
          x_inp = (x_inp<0) ? x_inp+Nx : ( (x_inp>Nx-1) ? x_inp-Nx : x_inp );
          y_inp = (y_inp<0) ? y_inp+Ny : ( (y_inp>Ny-1) ? y_inp-Ny : y_inp );
          z_inp = (z_inp<0) ? z_inp+Nz : ( (z_inp>Nz-1) ? z_inp-Nz : z_inp );
          
          // x,y,z coordinates for current thread
          Scalar3 pos_grid;
          pos_grid.x = gridh.x*x_inp - Ld2.x;
          pos_grid.y = gridh.y*y_inp - Ld2.y;
          pos_grid.z = gridh.z*z_inp - Ld2.z;

          // Shear the grid position 
          // !!! This only works for linear shear where the shear gradient is along y
          //     and the shear direction is along x
          pos_grid.x = pos_grid.x + box.getTiltFactorXY() * pos_grid.y;
          
          // Global index for current grid point
          int grid_idx = x_inp * Ny * Nz + y_inp * Nz + z_inp;
          
          // Distance from particle to grid node
          Scalar3 r = pos_grid - pos;
          r = box.minImage(r);
          Scalar rsq = r.x*r.x + r.y*r.y + r.z*r.z;
          
          // Spreading Factor
          Scalar Cfac = prefac * expf( -expfac * rsq );
          
          // Get velocity from reduction (THIS IS THE SLOW STEP):
          velocity[thread_offset] += Cfac * make_scalar3( gridX[grid_idx].x, gridY[grid_idx].x, gridZ[grid_idx].x );
        }
      }//kk
    }//jj
  }//ii

  // Intra-block reduction for the total particle velocity
  // (add contributions from all grid points)
  int offs = block_size;
  int offs_prev; 
  while (offs > 1)
  {
        offs_prev = offs; 
        offs = ( offs + 1 ) / 2;
    __syncthreads();
        if (thread_offset + offs < offs_prev)
          {
            velocity[thread_offset] += velocity[thread_offset + offs];
          }
        
  }
  
  // Write out to global memory
  if (thread_offset == 0){
    d_vel[idx] = make_scalar4(velocity[0].x, velocity[0].y, velocity[0].z, d_vel[idx].w);
  }
  
}

/*!
  Wrapper to drive all the kernel functions used to compute 
  the wave space part of Mobility ( Same Size Particles )

*/
/*! \param d_pos            positions of the particles, actually they are fetched on texture memory
    \param d_vel            particle velocity
    \param d_net_force      net forces on the particles
    \param group_size       size of the group, i.e. number of particles
    \param d_group_members  index array to global HOOMD tag on each particle
    \param box              array containing box dimensions
    \param xi               Ewald splitting parameter
    \param eta              Spectral splitting parameter
    \param ewald_cut        Cut-off distance for real-space interaction
    \param ewald_dr         Distance spacing using in computing the pre-tabulated tables
    \param ewald_n          Number of entries in the Ewald tables
    \param d_ewaldC         Pre-tabulated form of the real-space Ewald sum for the Velocity-Force coupling
    \param d_gridX          x-component of force moments projected onto grid
    \param d_gridY          y-component of force moments projected onto grid
    \param d_gridZ          z-component of force moments projected onto grid
    \param d_gridk          wave vector and scaling factor associated with each reciprocal grid node
    \param plan             Plan for cufft
    \param Nx               Number of grid/FFT nodes in x-direction
    \param Ny               Number of grid/FFT nodes in y-direction
    \param Nz               Number of grid/FFT nodes in z-direction
    \param d_n_neigh        list containing number of neighbors for each particle
    \param d_nlist          list containing neighbors of each particle
    \param nli              index into nlist
    \param NxNyNz           total number of grid/FFT nodes
    \param grid             block grid to use when launching kernels
    \param threads          number of threads per block for kernels
    \param gridBlockSize    number of threads per block
    \param gridNBlock       number of blocks
    \param P                number of nodes in support of each gaussian for k-space sum
    \param gridh            distance between grid nodes
*/
void gpu_stokes_Mwave_wrap( 
        Scalar4 *d_pos,
                          Scalar4 *d_vel,
                          Scalar4 *d_net_force,
        unsigned int *d_group_members,
        unsigned int group_size,
                          const BoxDim& box,
        Scalar xi,
        Scalar eta,
        Scalar4 *d_gridk,
        CUFFTCOMPLEX *d_gridX,
        CUFFTCOMPLEX *d_gridY,
        CUFFTCOMPLEX *d_gridZ,
        hipfftHandle plan,
        const int Nx,
        const int Ny,
        const int Nz,
        unsigned int NxNyNz,
        dim3 grid,
        dim3 threads,
        int gridBlockSize,
        int gridNBlock,
        const int P,
        Scalar3 gridh 
        ){
    
  // Spreading and contraction stuff
  dim3 Cgrid( group_size, 1, 1);
  int B = ( P < 10 ) ? P : 10;
  dim3 Cthreads(B, B, B);

  Scalar quadW = gridh.x * gridh.y * gridh.z;
  Scalar xisq = xi * xi;
  Scalar prefac = ( 2.0 * xisq / 3.1415926536 / eta ) * sqrtf( 2.0 * xisq / 3.1415926536 / eta );
  Scalar expfac = 2.0 * xisq / eta;
  
  // Reset the grid ( remove any previously distributed forces )
  gpu_stokes_ZeroGrid_kernel<<<gridNBlock,gridBlockSize>>>(d_gridX,NxNyNz);
  gpu_stokes_ZeroGrid_kernel<<<gridNBlock,gridBlockSize>>>(d_gridY,NxNyNz);
  gpu_stokes_ZeroGrid_kernel<<<gridNBlock,gridBlockSize>>>(d_gridZ,NxNyNz);
  
  // Spread forces onto grid
  gpu_stokes_Spread_kernel<<<Cgrid, Cthreads>>>( d_pos, d_net_force, d_gridX, d_gridY, d_gridZ, group_size, Nx, Ny, Nz, d_group_members, box, P, gridh, xi, eta, prefac, expfac );
  
  // Perform FFT on gridded forces
  hipfftExecC2C(plan, d_gridX, d_gridX, HIPFFT_FORWARD);
  hipfftExecC2C(plan, d_gridY, d_gridY, HIPFFT_FORWARD);
  hipfftExecC2C(plan, d_gridZ, d_gridZ, HIPFFT_FORWARD);
  
  // Apply wave space scaling to FFT'd forces
  gpu_stokes_Green_kernel<<<gridNBlock,gridBlockSize>>>( d_gridX, d_gridY, d_gridZ, d_gridk, NxNyNz);
  
  // Return rescaled forces to real space
  hipfftExecC2C(plan, d_gridX, d_gridX, HIPFFT_BACKWARD);
  hipfftExecC2C(plan, d_gridY, d_gridY, HIPFFT_BACKWARD);
  hipfftExecC2C(plan, d_gridZ, d_gridZ, HIPFFT_BACKWARD);
  
  // Evaluate contribution of grid velocities at particle centers
  gpu_stokes_Contract_kernel<<<Cgrid, Cthreads, (B*B*B+1)*sizeof(float3)>>>( d_pos, d_vel, d_gridX, d_gridY, d_gridZ, group_size, Nx, Ny, Nz, xi, eta, d_group_members, box, P, gridh, quadW*prefac, expfac );
 
}

// Add real space Ewald summation to velocity of each particle
// NLIST Method
/*! \param d_pos            positions of the particles, actually they are fetched on texture memory
    \param d_vel            particle velocity
    \param d_net_force      net forces on the particles
    \param group_size       size of the group, i.e. number of particles
    \param xi               Ewald splitting parameter
    \param d_ewaldC         Pre-tabulated form of the real-space Ewald sum for the Velocity-Force coupling
    \param ewald_cut        Cut-off distance for real-space interaction
    \param ewald_n          Number of entries in the Ewald tables
    \param ewald_dr         Distance spacing using in computing the pre-tabulated tables
    \param d_group_members  index array to global HOOMD tag on each particle
    \param box              array containing box dimensions
    \param d_n_neigh        list containing number of neighbors for each particle
    \param d_nlist          list containing neighbors of all particles
    \param d_headlist       list of particle offsets into d_nlist
*/
__global__ void gpu_stokes_Mreal_kernel(   
        Scalar4 *d_pos,
              Scalar4 *d_vel,
              Scalar4 *d_net_force,
              int group_size,
              Scalar xi,
              Scalar4 *d_ewaldC1, 
              Scalar self,
              Scalar ewald_cut,
              int ewald_n,
              Scalar ewald_dr,
              unsigned int *d_group_members,
              BoxDim box,
              const unsigned int *d_n_neigh,
                                const unsigned int *d_nlist,
                                const unsigned int *d_headlist
        ){
 
  // Index for current thread 
  int group_idx = blockDim.x * blockIdx.x + threadIdx.x;
  
  // Initialize contribution to velocity
  Scalar4 u = make_scalar4( 0.0, 0.0, 0.0, 0.0 );
  
  if (group_idx < group_size) {
    
    // Particle for this thread
    unsigned int idx = d_group_members[group_idx];
    
    // Number of neighbors for current particle
    unsigned int n_neigh = d_n_neigh[idx]; 
    unsigned int head_idx = d_headlist[idx];
    
    // Particle position and table ID
    Scalar4 posi = texFetchScalar4(d_pos, pos_tex, idx);
    
    // Self contribution
    Scalar4 F = d_net_force[idx];
    u = make_scalar4( self * F.x, self * F.y, self * F.z, 0.0 );
    
    // Minimum and maximum distance for pair calculation
    Scalar mindistSq = ewald_dr * ewald_dr;
    Scalar maxdistSq = ewald_cut * ewald_cut;
    
    for (int neigh_idx = 0; neigh_idx < n_neigh; neigh_idx++) {

      // Get index for current neightbor
      unsigned int cur_j = d_nlist[ head_idx + neigh_idx ];  
  
      // Position and size of neighbor particle
      Scalar4 posj = texFetchScalar4(d_pos, pos_tex, cur_j);
    
      // Distance vector between current particle and neighbor
      Scalar3 r = make_scalar3( posi.x - posj.x, posi.y - posj.y, posi.z - posj.z );
      r = box.minImage(r);
      Scalar distSqr = dot(r,r);
    
      // Add neighbor contribution if it is within the real space cutoff radius
      if ( ( distSqr < maxdistSq ) && ( distSqr >= mindistSq ) ) {
    
        // Need distance 
        Scalar dist = sqrtf( distSqr );
        
        // Force on neighbor particle
        Scalar4 Fj = d_net_force[cur_j];
      
        // Fetch relevant elements from textured table for real space interaction
        int r_ind = __scalar2int_rd( ewald_n * ( dist - ewald_dr ) / ( ewald_cut - ewald_dr ) );
        int offset = r_ind;
    
        Scalar4 tewaldC1 = texFetchScalar4(d_ewaldC1, tables1_tex, offset);
    
        // Linear interpolation of table
        Scalar fac = dist / ewald_dr - r_ind - Scalar(1.0);
    
        Scalar Imrr = tewaldC1.x + ( tewaldC1.z - tewaldC1.x ) * fac;
        Scalar rr = tewaldC1.y + ( tewaldC1.w - tewaldC1.y ) * fac;
    
        // Update velocity
        Scalar rdotf = ( r.x*Fj.x + r.y*Fj.y + r.z*Fj.z ) / distSqr;
    
        u.x += Imrr * Fj.x + ( rr - Imrr ) * rdotf * r.x;
        u.y += Imrr * Fj.y + ( rr - Imrr ) * rdotf * r.y;
        u.z += Imrr * Fj.z + ( rr - Imrr ) * rdotf * r.z;
    
      }
    
    }
    
    // Write to output
    d_vel[idx] = u;
  
  }    
}



/*!
  Wrap all the functions to compute U = M * F ( SAME SIZE PARTICLES )
  Drive GPU kernel functions

  d_vel = M * d_net_force

*/
/*! \param d_pos            positions of the particles, actually they are fetched on texture memory
    \param d_vel            particle velocity
    \param d_net_force      net forces on the particles
    \param group_size       size of the group, i.e. number of particles
    \param d_group_members  index array to global HOOMD tag on each particle
    \param box              array containing box dimensions
    \param xi               Ewald splitting parameter
    \param eta              Spectral splitting parameter
    \param ewald_cut        Cut-off distance for real-space interaction
    \param ewald_dr         Distance spacing using in computing the pre-tabulated tables
    \param ewald_n          Number of entries in the Ewald tables
    \param d_ewaldC         Pre-tabulated form of the real-space Ewald sum for the Velocity-Force coupling
    \param d_gridX          x-component of force moments projected onto grid
    \param d_gridY          y-component of force moments projected onto grid
    \param d_gridZ          z-component of force moments projected onto grid
    \param d_gridk          wave vector and scaling factor associated with each reciprocal grid node
    \param plan             Plan for cufft
    \param Nx               Number of grid/FFT nodes in x-direction
    \param Ny               Number of grid/FFT nodes in y-direction
    \param Nz               Number of grid/FFT nodes in z-direction
    \param d_n_neigh        list containing number of neighbors for each particle
    \param d_nlist          list containing neighbors of each particle
    \param nli              index into nlist
    \param NxNyNz           total number of grid/FFT nodes
    \param grid             block grid to use when launching kernels
    \param threads          number of threads per block for kernels
    \param gridBlockSize    number of threads per block
    \param gridNBlock       number of blocks
    \param P                number of nodes in support of each gaussian for k-space sum
    \param gridh            distance between grid nodes
*/
void gpu_stokes_Mobility_wrap( 
        Scalar4 *d_pos,
        Scalar4 *d_vel,
        Scalar4 *d_net_force,
        unsigned int *d_group_members,
        unsigned int group_size,
        const BoxDim& box,
        Scalar xi,
        Scalar eta,
        Scalar ewald_cut,
        Scalar ewald_dr,
        int ewald_n,
        Scalar4 *d_ewaldC1, 
        Scalar self,
        Scalar4 *d_gridk,
        CUFFTCOMPLEX *d_gridX,
        CUFFTCOMPLEX *d_gridY,
        CUFFTCOMPLEX *d_gridZ,
        hipfftHandle plan,
        const int Nx,
        const int Ny,
        const int Nz,
        const unsigned int *d_n_neigh,
        const unsigned int *d_nlist,
        const unsigned int *d_headlist,
        unsigned int NxNyNz,
        dim3 grid,
        dim3 threads,
        int gridBlockSize,
        int gridNBlock,
        const int P,
        Scalar3 gridh ){

  // Real and wave space velocity
  Scalar4 *d_vel1, *d_vel2;
  hipMalloc( &d_vel1, group_size*sizeof(Scalar4) );
  hipMalloc( &d_vel2, group_size*sizeof(Scalar4) );
  
  // Add the wave space contribution to the velocity
  gpu_stokes_Mwave_wrap( d_pos, d_vel1, d_net_force, d_group_members, group_size, box, xi, eta, d_gridk, d_gridX, d_gridY, d_gridZ, plan, Nx, Ny, Nz, NxNyNz, grid, threads, gridBlockSize, gridNBlock, P, gridh );
  
  // Add the real space contribution to the velocity
  //
  // Real space calculation takes care of self contributions
  gpu_stokes_Mreal_kernel<<<grid, threads>>>(d_pos, d_vel2, d_net_force, group_size, xi, d_ewaldC1, self, ewald_cut, ewald_n, ewald_dr, d_group_members, box, d_n_neigh, d_nlist, d_headlist );
  
  // Add real and wave space parts together
  gpu_stokes_LinearCombination_kernel<<<grid, threads>>>(d_vel1, d_vel2, d_vel, 1.0, 1.0, group_size, d_group_members);
  
  // Free memory
  hipFree(d_vel1);
  hipFree(d_vel2);
 
}


